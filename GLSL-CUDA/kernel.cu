#include "hip/hip_runtime.h"

#include <glad/glad.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <cuda_gl_interop.h>

hipGraphicsResource* cudaVBO;

float* vertices_dev;
float* Vel_dev;
int* head_dev;
int* lscl_dev;
float* Orts_dev;
float* radius_dev;
float* inter_dev;
int* L_dev;
int* c_dev;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ void addKernel(float* vel, float* orts, const int* size)
{
	int i = blockIdx.x  * blockDim.x + threadIdx.x;
	if (i < (*size)) {
		float hypot = sqrt(powf(vel[3 * i], 2) + powf(vel[3 * i + 1], 2) + powf(vel[3 * i + 2], 2));
		orts[3 * i] = vel[3 * i] / hypot;
		orts[3 * i + 1] = vel[3 * i + 1] / hypot;
		orts[3 * i + 2] = vel[3 * i + 2] / hypot;
	}
}

__global__ void nullKernel(int* head, const int* L)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < ((*L) * (*L) * (*L)))
		head[i] = -1;
}



__global__ void head_kernel(int* head, float* vertices, int* lscl, const float* radius, const float* inter, const int* L, const int* size)
{
	
	for (size_t i = 0; i < *size; i++) {
		size_t x1 = ((vertices[3 * i] + (*radius)) / (*inter));
		size_t y1 = ((vertices[3 * i + 1] + (*radius)) / (*inter));
		size_t z1 = ((vertices[3 * i + 2] + (*radius)) / (*inter));

		int c = x1 * ((*L) * (*L)) + y1 * (*L) + z1;

		lscl[i] = head[c];
		head[c] = i;
	}
}


__global__ void main_dev(float* vertices, float *Vel,int* head, int* lscl, float *Orts, const float* radius, const float* inter, const int* size, const int* L)
{


	int xc = blockIdx.x * blockDim.x + threadIdx.x;
	int yc = blockIdx.y * blockDim.y + threadIdx.y;
	int zc = blockIdx.z * blockDim.z + threadIdx.z;
	if (xc < (*L) && yc < (*L) && zc < (*L)) {
		int c = xc * (*L * *L) + yc * (*L) + zc;

		float sx = 0.;
		float sy = 0.;
		float sz = 0.;

		float Hypot, x, y, z, vx, vy, vz, x1, y1, z1, vx1, vy1, vz1;
		float SQRT, dt1, x2, y2, z2, Veln, Velx, Vely, Velz, Velx1, Vely1, Velz1, nx, ny, nz;
		float coef = 0.01;

		for (int xc1 = xc - 1; xc1 <= xc + 1; xc1++)
			for (int yc1 = yc - 1; yc1 <= yc + 1; yc1++)
				for (int zc1 = zc - 1; zc1 <= zc + 1; zc1++) {
					int c1 = ((xc + *L) % (*L)) * (*L * *L) + ((yc1 + *L) % (*L)) * (*L) + ((zc1 + *L) % (*L));
					int i = head[c];
					while (i != -1) {
						sx = 0.;
						sy = 0.;
						sz = 0.;
						int j = head[c1];
						while (j != -1) {
							if (i != j) {
								float dist = sqrt(pow(vertices[3 * (size_t)i] - vertices[3 * (size_t)j], 2) + pow(vertices[3 * (size_t)i + 1] - vertices[3 * (size_t)j + 1], 2) + pow(vertices[3 * (size_t)i + 2] - vertices[3 * (size_t)j + 2], 2));
								if (dist <= (*inter)) {
									sx += Orts[3 * j];
									sy += Orts[3 * j + 1];
									sz += Orts[3 * j + 2];
								}
							}
							j = lscl[j];
						}
					
				
	
						/*
						/*phi = glm::pi<float>() - 2 * glm::pi<float>() * ((float)rand() / (RAND_MAX));
						noise_z = 1. - 2. * ((float)rand() / (RAND_MAX));
						noise_x = sqrt(1 - pow(noise_z, 2)) * cos(phi);
						noise_y = sqrt(1 - pow(noise_z, 2)) * sin(phi);
						*/
						if (abs(sx) < 1e-4 && abs(sy) < 1e-4 && abs(sz) < 1e-4) {
							/*vx = Orts[3 * i];
							vy = Orts[3 * i + 1];
							vz = Orts[3 * i + 2];
							Hypot = sqrt(pow(vx, 2) + pow(vy, 2) + pow(vz, 2));
							Orts[3 * i] = vx / Hypot;
							Orts[3 * i + 1] = vy / Hypot;
							Orts[3 * i + 2] = vz / Hypot;*/
						}
						else {
							Hypot = sqrt(pow(sx, 2) + pow(sy, 2) + pow(sz, 2));
							Orts[3 * i] = sx / Hypot;
							Orts[3 * i + 1] = sy / Hypot;
							Orts[3 * i + 2] = sz / Hypot;
							//Hypot = sqrt(pow(Orts[3 * i], 2) + pow(Orts[3 * i + 1], 2) + pow(Orts[3 * i + 2], 2));
							//Orts[3 * i] = Orts[3 * i] / Hypot;
							//Orts[3 * i + 1] = Orts[3 * i + 1] / Hypot;
							//Orts[3 * i + 2] = Orts[3 * i + 2] / Hypot;
						}


						

						Hypot = sqrt(pow(Vel[3 * i], 2) + pow(Vel[3 * i + 1], 2) + pow(Vel[3 * i + 2], 2));

						x = vertices[3 * (size_t)i];
						y = vertices[3 * (size_t)i + 1];
						z = vertices[3 * (size_t)i + 2];

						vx = Orts[3 * i] * Hypot;
						vy = Orts[3 * i + 1] * Hypot;
						vz = Orts[3 * i + 2] * Hypot;

						x1 = x + vx * coef;
						y1 = y + vy * coef;
						z1 = z + vz * coef;

						if (pow(x1, 2) + pow(y1, 2) + pow(z1, 2) >= pow((*radius), 2))
						{
							SQRT = pow((x * vx + y * vy + z * vz) / (pow(vx, 2) + pow(vy, 2) + pow(vz, 2)), 2) + (pow((*radius), 2) - pow(x, 2) - pow(y, 2) - pow(z, 2)) / (pow(vx, 2) + pow(vy, 2) + pow(vz, 2));
							dt1 = fabs(sqrt(fabs(SQRT)) - (x * vx + y * vy + z * vz) / (pow(vx, 2) + pow(vy, 2) + pow(vz, 2)));

							x2 = x + vx * dt1;
							y2 = y + vy * dt1;
							z2 = z + vz * dt1;

							Velx = vx;
							Vely = vy;
							Velz = vz;

							nx = -x2 / (*radius);
							ny = -y2 / (*radius);
							nz = -z2 / (*radius);

							Veln = Velx * nx + Vely * ny + Velz * nz;
							Velx1 = Velx - 2 * nx * Veln;
							Vely1 = Vely - 2 * ny * Veln;
							Velz1 = Velz - 2 * nz * Veln;

							vx = Velx1;
							vy = Vely1;
							vz = Velz1;

							x = x2 + vx * (coef - dt1);
							y = y2 + vy * (coef - dt1);
							z = z2 + vz * (coef - dt1);

						}
						else {
							x = x1;
							y = y1;
							z = z1;
						}
						vertices[3 * (size_t)i] = x;
						vertices[3 * (size_t)i + 1] = y;
						vertices[3 * (size_t)i + 2] = z;

						Vel[3 * (size_t)i] = vx;
						Vel[3 * (size_t)i + 1] = vy;
						Vel[3 * (size_t)i + 2] = vz;

						i = lscl[i];
					}



				}
	}
}


extern void cudaInit(unsigned int VBO, float* vertices, float* Vel, int* head, int* lscl, float* Orts, const float radius, const float inter, const int size, const int L)
{
	hipGraphicsGLRegisterBuffer(&cudaVBO, VBO, hipGraphicsRegisterFlagsWriteDiscard);

	hipMalloc(&vertices_dev, size * sizeof(float) * 3);
	hipGraphicsMapResources(1, &cudaVBO, NULL);
	hipGraphicsResourceGetMappedPointer((void**)&vertices_dev, NULL, cudaVBO);
	hipMemcpy(vertices_dev, vertices, size * sizeof(float) * 3, hipMemcpyHostToDevice);
	
	hipMalloc(&Vel_dev, size * sizeof(float) * 3);
	hipMemcpy(Vel_dev, Vel, size * sizeof(float) * 3, hipMemcpyHostToDevice);

	hipMalloc(&head_dev, (L * L * L * L) * sizeof(int));
	hipMemcpy(head_dev, head, (L * L * L) * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&lscl_dev, size * sizeof(int));
	hipMemcpy(lscl_dev, lscl, size * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&Orts_dev, size * sizeof(float) * 3);
	hipMemcpy(Orts_dev, Orts, size * sizeof(float) * 3, hipMemcpyHostToDevice);

	hipMalloc(&radius_dev, sizeof(float));
	hipMemcpy(radius_dev, &radius, sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&inter_dev, sizeof(float));
	hipMemcpy(inter_dev, &inter, sizeof(float), hipMemcpyHostToDevice);
	
	hipMalloc(&L_dev, sizeof(int));
	hipMemcpy(L_dev, &L, sizeof(int), hipMemcpyHostToDevice);
	
	hipMalloc(&c_dev, sizeof(int));
	hipMemcpy(c_dev, &size, sizeof(int), hipMemcpyHostToDevice);
	
}

extern void kernelUpdate(const int size, const int L)
{
	
	
	dim3 threadIn(512);
	dim3 blockIn(size / threadIn.x + 1);
	addKernel << <blockIn, threadIn >>> (Vel_dev, Orts_dev, c_dev);
	gpuErrchk(hipPeekAtLastError());
	dim3 threadNull(512);
	int L3 = L * L * L;
	dim3 BlockNull(L3 / 512 + 1);
	nullKernel << <BlockNull, threadNull >> > (head_dev, L_dev);
	gpuErrchk(hipPeekAtLastError());
	head_kernel << <1, 1 >> > (head_dev, vertices_dev, lscl_dev, radius_dev, inter_dev, L_dev, c_dev);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipPeekAtLastError());
	dim3 thread(8, 8, 8);
	dim3 blocks(L / thread.x + 1, L / thread.y + 1, L / thread.z + 1);
	main_dev << <blocks, thread >> > (vertices_dev, Vel_dev, head_dev, lscl_dev, Orts_dev, radius_dev, inter_dev, c_dev, L_dev);
	
	gpuErrchk(hipPeekAtLastError());
	
	
	
}

extern void kernelTerminate()
{
	hipGraphicsUnmapResources(1, &cudaVBO, NULL);
	hipFree(vertices_dev);
	hipFree(Vel_dev);
	hipFree(Orts_dev);
	hipFree(head_dev);
	hipFree(lscl_dev);
	hipFree(radius_dev);
	hipFree(inter_dev);
	hipFree(L_dev);
	hipFree(c_dev);
}